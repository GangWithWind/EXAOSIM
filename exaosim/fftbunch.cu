#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
using namespace std;
#define CHANNEL_NUM  10 //通道数、FFT次数
const int dataH = 512; //图像高度
const int dataW = 512;  //图像宽度
hipfftHandle fftplanfwd; //创建句柄


__global__ void SetFFTInput(hipfftComplex* input, int H, int W, int Nb)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int ix = 0, ib = 0;
    if (i < H * Nb){
        ix = i % H;
        ib = i / H + 10;
        if(ix < ib){
            for(int j = 0; j < ib; j++){
                input[i * W + j].y = 1;
            }
        }

    }
}


int main(void){
 /* 开辟主机端的内存空间 */
 printf("文件名planmany_cuda31.cu...\n");
 printf("分配CPU内存空间...\n");
 hipfftComplex *h_Data = (hipfftComplex*)malloc(dataH*CHANNEL_NUM*dataW* sizeof(hipfftComplex));//可用hipHostMalloc设置
 hipfftComplex *h_resultFFT = (hipfftComplex*)malloc(dataH*CHANNEL_NUM*dataW* sizeof(hipfftComplex));
 /* 开辟设备端的内存空间 */
 printf("分配GPU内存空间...\n");
 /* 定义设备端的内存空间 */
 hipfftComplex *d_Data;//device表示GPU内存，存储从cpu拷贝到GPU的数据
 hipfftComplex *fd_Data;//device表示GPU内存,R2C后存入hipfftComplex类型数据
 checkCudaErrors(hipMalloc((void**)&d_Data, dataH*CHANNEL_NUM*dataW* sizeof(hipfftComplex)));
 checkCudaErrors(hipMemset(d_Data, 0, dataH*CHANNEL_NUM * dataW* sizeof(hipfftComplex))); // 初始为0
 checkCudaErrors(hipMalloc((void**)&fd_Data, dataH*CHANNEL_NUM*dataW* sizeof(hipfftComplex))); // 开辟R2C后的设备内存
 checkCudaErrors(hipMemset(fd_Data, 0, dataH*CHANNEL_NUM*dataW* sizeof(hipfftComplex))); // 初始为0
 //随机初始化测试数据
 printf("初始化测试数据...\n");
 for (int k = 0; k < CHANNEL_NUM; k++){
    for (int i = 0; i < dataH; i++){
        for (int j = 0; j < dataW; j++){
            h_Data[(i + k * dataH) * dataW + j].x = 0;//float(rand()%255);
            h_Data[(i + k * dataH) * dataW + j].y = 0;//float(rand()%255);
            if(i < (4 + k) && j < (4 + k)){
                h_Data[(i + k * dataH)*dataW + j].x = 1;//float(rand()%255);
            }

        }
    }
}

int threadsPerBlock = 256;
int blocksPerGrid =
        (dataH * CHANNEL_NUM + threadsPerBlock - 1) / threadsPerBlock;
SetFFTInput<<<blocksPerGrid, threadsPerBlock>>>(d_Data, dataH, dataW, CHANNEL_NUM);
hipDeviceSynchronize();

 //使用event计算时间
 float time_elapsed = 0;
 hipEvent_t start, stop;
 hipEventCreate(&start);    //创建Event
 hipEventCreate(&stop);
 const int rank = 2;//维数
 int n[rank] = { dataH, dataW };//n*m
 int*inembed = n;//输入的数组sizecudaMemcpyHostToDevice
 int istride = 1;//数组内数据连续，为1
 int idist = n[0] * n[1];//1个数组的内存大小
 int*onembed = n;//输出是一个数组的size
 int ostride = 1;//每点DFT后数据连续则为1
 int odist = n[0] * n[1];//输出第一个数组与第二个数组的距离，即两个数组的首元素的距离
 int batch = CHANNEL_NUM;//批量处理的批数
 //采用hipfftPlanMany方法
 checkCudaErrors(
  hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));//针对多信号同时进行FFT
 printf("拷贝CPU数据到GPU中...\n");
//  checkCudaErrors(
//   hipMemcpy(d_Data, h_Data, dataW * dataH*CHANNEL_NUM * sizeof(hipfftComplex), hipMemcpyHostToDevice));
//  //printf("执行R2C-FFT...\n");
 printf("开始计时...\n");
 hipEventRecord(start, 0);    //记录当前时间
 checkCudaErrors(
  hipfftExecC2C(fftplanfwd, d_Data, fd_Data, HIPFFT_FORWARD));
 hipEventRecord(stop, 0);    //记录当前时间
 hipEventSynchronize(start);    //Waits for an event to complete.
 hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
 hipEventElapsedTime(&time_eaAlapsed, start, stop);    //计算时间差
 hipDeviceSynchronize();
 printf("拷贝GPU数据返回到CPU中...\n");
 checkCudaErrors(
  hipMemcpy(h_resultFFT, fd_Data, dataW *dataH*CHANNEL_NUM * sizeof(hipfftComplex), hipMemcpyDeviceToHost));//将fft后的数据拷贝回主机
 printf("显示返回到CPU中的数据...\n");

FILE *fp;
fp = fopen("test.txt", "w");

for (int i = 0; i < dataH*CHANNEL_NUM*dataW; i++){
    fprintf(fp, "%.10f\n", h_resultFFT[i].x*h_resultFFT[i].x + h_resultFFT[i].y*h_resultFFT[i].y);
    //  cout << "h_resultFFT[" << i << "]=" << h_resultFFT[i].x << " + " << h_resultFFT[i].y << " i" << endl;
}
fclose(fp);

 hipEventDestroy(start);    //destory the event
 hipEventDestroy(stop);
 printf("执行时间：%f(ms)\n", time_elapsed);
 /* 销毁句柄 */
 checkCudaErrors(hipfftDestroy(fftplanfwd));
 /* 释放设备空间 */
 checkCudaErrors(hipFree(d_Data));
 checkCudaErrors(hipFree(fd_Data));
 free(h_Data);
 free(h_resultFFT);
 return 0;
}